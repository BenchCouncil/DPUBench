#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021 NVIDIA CORPORATION & AFFILIATES, ALL RIGHTS RESERVED.
 *
 * This software product is a proprietary product of NVIDIA CORPORATION &
 * AFFILIATES (the "Company") and all right, title, and interest in and to the
 * software product, including all associated intellectual property rights, are
 * and shall remain exclusively with the Company.
 *
 * This software product is governed by the End User License Agreement
 * provided with the software product.
 *
 */

#include "dns_filter_kernel.h"
/* disable gnu_printf  warnings */
#define gnu_printf printf
#include <rte_ethdev.h>

extern "C" void print_l4_header_gpu_wrapper(struct rte_ipv4_hdr * gpu_ipv4_hdr, uint8_t ip_hdr_len);

__global__ void
print_l4_header_gpu(struct rte_ipv4_hdr * gpu_ipv4_hdr, uint8_t ip_hdr_len)
{
        uint8_t *l4_hdr;
        const struct rte_tcp_hdr *tcp_hdr;
        const struct rte_udp_hdr *udp_hdr;
	uint16_t dst_port;
	uint16_t src_port;

        l4_hdr = (typeof(l4_hdr))gpu_ipv4_hdr + ip_hdr_len;

        switch (gpu_ipv4_hdr->next_proto_id) {
        case IPPROTO_UDP:
                udp_hdr = (typeof(udp_hdr))l4_hdr;
                /* conversion from little endian to big endian */
		dst_port = (udp_hdr->dst_port)>>8 | (udp_hdr->dst_port)<<8;
		src_port = (udp_hdr->src_port)>>8 | (udp_hdr->src_port)<<8;

                printf("UDP- DPORT %u, SPORT %u\n", dst_port, src_port);
        break;

        case IPPROTO_TCP:
                tcp_hdr = (typeof(tcp_hdr))l4_hdr;
                /* conversion from little endian to big endian */
 		dst_port = (tcp_hdr->dst_port)>>8 | (tcp_hdr->dst_port)<<8;
                src_port = (tcp_hdr->src_port)>>8 | (tcp_hdr->src_port)<<8;

                printf("TCP- DPORT %u, SPORT %u\n", dst_port, src_port);
        break;

        default:
                printf("Unsupported L4 protocol!\n");
        }
}

void print_l4_header_gpu_wrapper(struct rte_ipv4_hdr * gpu_ipv4_hdr, uint8_t ip_hdr_len){
        printf("CUDA kernel launch for printing a packet received\n");
	print_l4_header_gpu<<<1,1>>>(gpu_ipv4_hdr, ip_hdr_len);
	hipDeviceSynchronize();
}
